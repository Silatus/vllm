#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <cmath>

#include "../../dispatch_utils.h"
#include "../../reduction_utils.cuh"


static inline __device__ int8_t float_to_int8_rn(float x) {
#ifdef USE_ROCM
  static const float i8_min =
      static_cast<float>(std::numeric_limits<int8_t>::min());
  static const float i8_max =
      static_cast<float>(std::numeric_limits<int8_t>::max());
  // round
  float dst = std::nearbyint(x);
  // saturate
  dst = std::clamp(dst, i8_min, i8_max);
  return static_cast<int8_t>(dst);
#else
  // CUDA path
  uint32_t dst;
  asm volatile("cvt.rni.sat.s8.f32 %0, %1;" : "=r"(dst) : "f"(x));
  return reinterpret_cast<const int8_t&>(dst);
#endif
}

namespace vllm {

template <typename scalar_t, typename scale_type>
__global__ void static_scaled_int8_quant_kernel(
    const scalar_t* __restrict__ input, int8_t* __restrict__ out,
    scale_type scale, const int hidden_size) {
  const int tid = threadIdx.x;
  const int token_idx = blockIdx.x;

  for (int i = tid; i < hidden_size; i += blockDim.x) {
    out[token_idx * hidden_size + i] =
        float_to_int8_rn(((float)input[token_idx * hidden_size + i]) / scale);
  }
}


template <typename scalar_t, typename scale_type>
__global__ void dynamic_scaled_int8_quant_kernel(
  const scalar_t* __restrict__ input,
  int8_t* __restrict__ out,
  scale_type scale,
  const int hidden_size) {

  const int tid = threadIdx.x;
  const int token_idx = blockIdx.x;

  float amax_val = 0.0f;
  const float zero = 0.0f;

  for (int i = tid; i < hidden_size; i += blockDim.x) {
    float val = (float)input[token_idx * hidden_size + i];
    val = val > zero ? val : -val;
    if (val > amax_val)
      amax_val = val;
  }

  __shared__ float s_amax;
  const float block_amax_val = blockReduceMax(amax_val);
  if (tid == 0) {
    s_amax = block_amax_val;
    scale[token_idx] = block_amax_val / 127.0f;
  }
  __syncthreads();

  float tmp_scale = 127.0f / s_amax;
  for (int i = tid; i < hidden_size; i += blockDim.x) {
    out[token_idx * hidden_size + i] =
        float_to_int8_rn(((float)input[token_idx * hidden_size + i]) * tmp_scale);
  }
}



void static_scaled_int8_quant(torch::Tensor& out,    // [..., hidden_size]
                              torch::Tensor& input,  // [..., hidden_size]
                              float scale) {
  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(out.is_contiguous());
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;
  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "static_scaled_int8_quant_kernel", [&] {
        vllm::static_scaled_int8_quant_kernel<scalar_t, float>
            <<<grid, block, 0, stream>>>(input.data_ptr<scalar_t>(),
                                         out.data_ptr<int8_t>(), scale,
                                         hidden_size);
      });
}


void dynamic_scaled_int8_quant(
  torch::Tensor& out,   // [..., hidden_size]
  torch::Tensor& input, // [..., hidden_size]
  torch::Tensor& scales) {
  assert(input.is_contiguous());
  assert(out.is_contiguous());
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;
  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "dynamic_scaled_int8_quant_kernel", [&] {
    vllm::dynamic_scaled_int8_quant_kernel<scalar_t, float*><<<grid, block, 0, stream>>>(
      input.data_ptr<scalar_t>(),
      out.data_ptr<int8_t>(),
      scales.data_ptr<float>(),
      hidden_size);
  });
}

