#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include "../../dispatch_utils.h"

static inline __device__ int8_t float_to_int8_rn(float x) {
  static constexpr float dt_min = static_cast<float>(std::numeric_limits<int8_t>::min());
  static constexpr float dt_max = static_cast<float>(std::numeric_limits<int8_t>::max());
  // round
  float dst = round(x);
  // saturate
  dst = std::clamp(dst, dt_min, dt_max);
  return static_cast<int8_t>(dst);
}

namespace vllm {

template <typename scalar_t, typename scale_type>
__global__ void static_scaled_int8_quant_kernel(
    const scalar_t* __restrict__ input, int8_t* __restrict__ out,
    scale_type scale, const int hidden_size) {
  const int tid = threadIdx.x;
  const int token_idx = blockIdx.x;

  for (int i = tid; i < hidden_size; i += blockDim.x) {
    out[token_idx * hidden_size + i] =
        float_to_int8_rn(((float)input[token_idx * hidden_size + i]) / scale);
  }
}
}  // namespace vllm

void static_scaled_int8_quant(torch::Tensor& out,    // [..., hidden_size]
                              torch::Tensor& input,  // [..., hidden_size]
                              float scale) {
  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(out.is_contiguous());
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;
  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "static_scaled_int8_quant_kernel", [&] {
        vllm::static_scaled_int8_quant_kernel<scalar_t, float>
            <<<grid, block, 0, stream>>>(input.data_ptr<scalar_t>(),
                                         out.data_ptr<int8_t>(), scale,
                                         hidden_size);
      });
}
