#ifdef USE_ROCM
  #include <hip/hip_runtime.h>
  #include <hip/hip_runtime_api.h>
#else
  
#include <hip/hip_runtime.h>
#include <cudaTypedefs.h>
#endif

// For querying the cuda version that the custom op library is compiled with
int64_t get_cuda_version()
{
#if defined CUDA_VERSION
    return CUDA_VERSION;
#else
    return -1;
#endif
}

int64_t get_device_attribute(int64_t attribute, int64_t device_id) {
  int device, value;
  if (device_id < 0) {
    hipGetDevice(&device);
  } else {
    device = device_id;
  }
  hipDeviceGetAttribute(&value, static_cast<hipDeviceAttribute_t>(attribute),
                         device);
  return value;
}

int64_t get_max_shared_memory_per_block_device_attribute(int64_t device_id) {
  int64_t attribute;
  // https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__TYPES.html
  // cudaDevAttrMaxSharedMemoryPerBlockOptin = 97 if not is_hip() else 74

#ifdef USE_ROCM
  attribute = hipDeviceAttributeMaxSharedMemoryPerBlock;
#else
  attribute = hipDeviceAttributeSharedMemPerBlockOptin;
#endif

  return get_device_attribute(attribute, device_id);
}
